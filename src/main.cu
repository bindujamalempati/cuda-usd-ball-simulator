#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include "usd_loader.hpp"
#include <vector>
#include <iostream>

int main() {
    std::vector<float3> positions, velocities;
    loadUSD("assets/scene.usd", positions, velocities);

    int N = positions.size();
    float3 *d_pos, *d_vel;
    hipMalloc(&d_pos, N * sizeof(float3));
    hipMalloc(&d_vel, N * sizeof(float3));
    hipMemcpy(d_pos, positions.data(), N * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_vel, velocities.data(), N * sizeof(float3), hipMemcpyHostToDevice);

    float dt = 0.016f;
    for (int frame = 0; frame < 50; ++frame) {
        simulate<<<(N+255)/256, 256>>>(d_pos, d_vel, N, dt);
        hipDeviceSynchronize();
    }

    hipMemcpy(positions.data(), d_pos, N * sizeof(float3), hipMemcpyDeviceToHost);
    hipFree(d_pos); hipFree(d_vel);
    std::cout << "Simulation done. Write frames using Python usd_writer.py\n";
    return 0;
}