#include "hip/hip_runtime.h"
#include "simulation.cuh"

__global__ void simulate(float3* positions, float3* velocities, int numObjects, float dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numObjects) {
        velocities[i].y -= 9.8f * dt;
        positions[i].x += velocities[i].x * dt;
        positions[i].y += velocities[i].y * dt;
        positions[i].z += velocities[i].z * dt;

        float radius = 0.5f;
        if (positions[i].y < radius) {
            positions[i].y = radius;
            velocities[i].y *= -0.8f;
        }
    }
}